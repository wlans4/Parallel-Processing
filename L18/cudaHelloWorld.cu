#include <stdlib.h>
#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void kernelHelloWorld(){



	int thread = threadIdx.x; //local thread number in a block
	int block = blockIdx.x;   //block number
	printf("Hello World from thread %d of block %d!\n", thread, block);


}


int main(int argc, char** argv){


	int Nblocks = 10; //number of blocks 
	int Nthreads = 3; //number of threads per blocks



	//run the function 'kernelHelloWorld on the DEVICE
	kernelHelloWorld <<< Nblocks, Nthreads >>> ();


	//Wait for the DEVICE function to complete before program finishes
	hipDeviceSynchronize();
	return 0;


}

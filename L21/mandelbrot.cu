#include "hip/hip_runtime.h"
/* 

To compile:

   nvcc -O3 -o mandelbrot mandelbrot.c png_util.c -I. -lpng -lm -fopenmp

Or just type:

   module load gcc
   make

To create an image with 4096 x 4096 pixels (last argument will be used to set number of threads):

    ./mandelbrot 4096 4096 1

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "png_util.h"
#include "hip/hip_runtime.h"

#define MXITER 1000

typedef struct {
  
  double r;
  double i;
  
}complex_t;

// return iterations before z leaves mandelbrot set for given c
__device__  int testpoint(complex_t c){
  
  int iter;

  complex_t z;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){
    
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return iter;
    }
  }
  
  
  return iter;
  
}

// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array

__global__ void  mandelbrot(int Nre, int Nim, complex_t cmin, complex_t cmax, float *count){ 
  int n,m;

  complex_t c;

  double dr = (cmax.r-cmin.r)/(Nre-1);
  double di = (cmax.i-cmin.i)/(Nim-1);;

  n = threadIdx.x + blockIdx.x * blockDim.x;
  m = threadIdx.y + blockIdx.y * blockDim.y;
	

  c.r = cmin.r + dr*m;
  c.i = cmin.i + di*n;
  count[m+n] = testpoint(c);
   

}

int main(int argc, char **argv){

  // to create a 4096x4096 pixel image [ last argument is placeholder for number of threads ] 
  // usage: ./mandelbrot 4096 4096 1  
  int Nre = atoi(argv[1]);
  int Nim = atoi(argv[2]);
  int Nthreads = atoi(argv[3]);

  
   float *d_a;
  //2D Block
  int NBlocksx = Nre;
  int NBlocksy = Nim;
  int NBlocksz = 1;
  //Num Blocks
  int NGridsx = 1;
  int NGridsy = 1;
  int NGridsz = 1;
  hipMalloc(&d_a, Nre*Nim*sizeof(float));
  dim3 B(NBlocksx, NBlocksy, NBlocksz); //2D threads
  dim3 G(NGridsx, NGridsy, NGridsz); //grid of threads
  
  // storage for the iteration counts
  float *count = (float*) malloc(Nre*Nim*sizeof(float));
  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;

  complex_t cmin; 
  complex_t cmax;

  cmin.r = centRe - 0.5*diam;
  cmax.r = centRe + 0.5*diam;
  cmin.i = centIm - 0.5*diam;
  cmax.i = centIm + 0.5*diam;

  clock_t start = clock(); //start time in CPU cycles

  // compute mandelbrot set
  mandelbrot <<<G,B>>> (Nre, Nim, cmin, cmax, count); 
  
  clock_t end = clock(); //start time in CPU cycles
  printf("MEMCPY\n"); 
  hipMemcpy(d_a, count, Nre*Nim*sizeof(float), hipMemcpyHostToDevice);
  // print elapsed time
  printf("elapsed = %f\n", ((double)(end-start))/CLOCKS_PER_SEC);

  // output mandelbrot to png format image
  FILE *fp = fopen("mandelbrot.png", "w");

  printf("Printing mandelbrot.png...");
 write_hot_png(fp, Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  

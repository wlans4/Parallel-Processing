#include "hip/hip_runtime.h"
/* 

To compile:

   nvcc -O3 -o mandelbrot mandelbrot.c png_util.c -I. -lpng -lm -fopenmp

Or just type:

   module load gcc
   make

To create an image with 4096 x 4096 pixels (last argument will be used to set number of threads):

    ./mandelbrot 4096 4096 1

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "png_util.h"

#include "hip/hip_runtime.h"

// Q2a: add include for CUDA header file here:

#define MXITER 1000

typedef struct {
  
  double r;
  double i;
  
}complex_t;

// return iterations before z leaves mandelbrot set for given c
//ADD __DEVICE__ to make visible to gpu
__device__  int testpoint(complex_t c){
  
  int iter;

  complex_t z;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){
    
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return iter;
    }
  }
  
  
  return iter;
  
}

// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array

// Q2c: transform this function into a CUDA kernel
__global__ void  mandelbrot(int Nre, int Nim, complex_t cmin, complex_t cmax, float *count){ 
  int n,m;

  complex_t c;

  double dr = (cmax.r-cmin.r)/(Nre-1);
  double di = (cmax.i-cmin.i)/(Nim-1);;

	n = threadIdx.x + blockIdx.x * blockDim.x;
	m = threadIdx.y + blockIdx.y * blockDim.y;
	
//  for(n=0;n<Nim;++n){
  //  for(m=0;m<Nre;++m){
      c.r = cmin.r + dr*m;
      c.i = cmin.i + di*n;
      //m+n*Nre
      count[m+n] = testpoint(c);
      
  //  }
//  }

}

int main(int argc, char **argv){

  // to create a 4096x4096 pixel image [ last argument is placeholder for number of threads ] 
  // usage: ./mandelbrot 4096 4096 1  
  
  printf("TEST\n");
  int Nre = atoi(argv[1]);
printf("Test2\n");
  int Nim = atoi(argv[2]);
printf("Test3\n");
  int Nthreads = atoi(argv[3]);

printf("testblahg\n");
  
  // Q2b: set the number of threads per block and the number of blocks here: 
  
  
   float *d_a;
printf("testblah2\n");
  //2D Block
  int NBlocksx = Nre;
  int NBlocksy = Nim;
  int NBlocksz = 1;
 printf("Test1.5\n"); 
  //Num Blocks
  int NGridsx = 1;
  int NGridsy = 1;
  int NGridsz = 1;
 printf("Test1.75\n"); 
 printf("Test2\n"); 
  hipMalloc(&d_a, Nre*Nim*sizeof(float));
  printf("Test3\n");
  dim3 B(NBlocksx, NBlocksy, NBlocksz); //2D threads
  dim3 G(NGridsx, NGridsy, NGridsz); //grid of threads
  
  // storage for the iteration counts
  float *count = (float*) malloc(Nre*Nim*sizeof(float));
  
  
  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;

  complex_t cmin; 
  complex_t cmax;

  cmin.r = centRe - 0.5*diam;
  cmax.r = centRe + 0.5*diam;
  cmin.i = centIm - 0.5*diam;
  cmax.i = centIm + 0.5*diam;

  clock_t start = clock(); //start time in CPU cycles

  // compute mandelbrot set
  mandelbrot <<<G,B>>> (Nre, Nim, cmin, cmax, count); 
  
  clock_t end = clock(); //start time in CPU cycles
  printf("MEMCPY\n"); 
  hipMemcpy(d_a, count, Nre*Nim*sizeof(float), hipMemcpyHostToDevice);
  // print elapsed time
  printf("elapsed = %f\n", ((double)(end-start))/CLOCKS_PER_SEC);

  // output mandelbrot to png format image
  FILE *fp = fopen("mandelbrot.png", "w");

  printf("Printing mandelbrot.png...");
 // write_hot_png(fp, Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"




__device__ unsigned int DeviceModprod(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__  unsigned int DeviceModExp(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = DeviceModprod(aExpb, z, p);
    z = DeviceModprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}


__global__ void findKey(unsigned int x, unsigned int p, unsigned int g, unsigned int h, unsigned int* result){

	int i = 0;
      if (DeviceModExp(g, i+1, p) == h) {
        printf("Secret key found! x = %u \n", i+1);
        *result = i + 1;
	i++;
      }
}

int main (int argc, char **argv) {
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  //Read in from public_key.txt
  FILE *file = fopen("public_key.txt", "r");
  if (file == NULL){
        printf("ERROR: public_key.txt does not exist\n");
        return -1;
  }

  fscanf(file, "%d %d %d %d", &n, &p, &g, &h);
  printf("Read in public_key.txt\n");

  file = fopen("message.txt", "r");
  if (file == NULL){
        printf("ERROR: message.txt does not exist\n");
        return -1;
  }

  fscanf(file, "%d", &Nints);
  unsigned int* ints = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  for (int i = 0; i < Nints - 1; i++){
        fscanf(file, "%u", (ints + i));
  }
  printf("Read in cyphertexts from messages.txt\n");


  int Nblocks = 1;
  int Nthreads = 1;
  unsigned int* h_count = (unsigned int*) malloc(1*sizeof(unsigned int));;
  unsigned int* d_count;
  hipMalloc(&d_count, sizeof(unsigned int));
  // find the secret key
  double startTime = clock();
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    findKey <<< Nblocks, Nthreads>>> (x, p, g, h, d_count);
      }
  	
    hipMemcpy(h_count, d_count, sizeof(unsigned int), hipMemcpyHostToDevice);
    double endTime = clock();
    printf("The secret key is %u\n", *h_count);
    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }






#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "functions.h"
#include "hip/hip_runtime.h"
#include "functions.c"
__device__ unsigned int deviceModProd(unsigned int a, unsigned int b, unsigned int p){
	unsigned int za = a;
	unsigned int ab = 0;
	

	while (b > 0){
		if (b%2 == 1) ab = (ab + za) % p;
		za = (2 * za) % p;
		b /= 2;
	}

	return ab;
}


__device__ unsigned int deviceModExp(unsigned int a, unsigned int b, unsigned int p){

	unsigned int z = a;
	unsigned int aExpb = 1;

	while (b > 0){
		if (b%2 == 1) aExpb = deviceModProd(aExpb, z, p);
		z = deviceModProd(z, z, p);
		b /= 2;
	}
	return aExpb;

}

__global__ void find(unsigned int p, unsigned int g, unsigned int h, unsigned int* result){

	unsigned int x = (unsigned int)(threadIdx.x + blockIdx.x*blockDim.x);
	unsigned int y = (unsigned int)(threadIdx.y + blockIdx.y*blockDim.y);

	unsigned int i = y*blockDim.x * gridDim.x + x;
	if (i < p){
		if (deviceModExp(g, i + 1, p) == h){
			*result = i + 1;
		}

	}
}



int main (int argc, char **argv) {

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;
  unsigned int Nchars;
  
  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u", &x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  //Read in from public_key.txt
  FILE *file = fopen("bonus_public_key.txt", "r");
  if (file == NULL){
	printf("ERROR: bonus_public_key.txt does not exist\n");
	return -1;
  }

  fscanf(file, "%d %d %d %d", &n, &p, &g, &h);
  printf("Read in public_key.txt\n");
  fclose(file);
  file = fopen("bonus_message.txt", "r");
  if (file == NULL){
	printf("ERROR: bonus_message.txt does not exist\n");
	return -1;
  }

  fscanf(file, "%u",&Nints);
  unsigned int* Z = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  unsigned int* a = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  for (int i = 0; i < Nints; i++){
	fscanf(file, "%u %u\n", &Z[i], &a[i]);
  }
  fclose(file);
  Nchars = Nints*(n-1)/8;





  unsigned int* h_x = (unsigned int*) malloc(sizeof(unsigned int));
  *h_x = 0;
  unsigned int* d_x;
  hipMalloc(&d_x, sizeof(unsigned int));
  dim3 B(32, 32, 1);
  int N = (n - 9)/2;
  if (N < 0){
	N = 0;
  }
  N = 1 << N;
  dim3 G(N, N, 1);

  printf("Read in cyphertexts from messages.txt\n");
  double startTime = clock();
 find  <<< G, B >>> (p, g, h, d_x);
  hipDeviceSynchronize();   
  double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

	
    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  


hipMemcpy(h_x, d_x, sizeof(unsigned int), hipMemcpyDeviceToHost);
printf("The secret key is %u\n", *h_x);

printf("The decrypted message is:\n");
        unsigned char* message = (unsigned char*) malloc(100*sizeof(unsigned char));
        ElGamalDecrypt(Z, a, Nints, p, *h_x);
        convertZToString(Z, Nints, message, Nchars);
        printf("\"%s\"\n", message);
        printf("\n");
	free(h_x);
	
}
